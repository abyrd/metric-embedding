
#include <hip/hip_runtime.h>
// CUDA kernels for embedding shortest path metric into normed vector space

// Calculate all pairs shortest path.
// after Okuyama, Ino, and Hagihara 2008.
__global__ void scatter (int nv, int *vertex, int *edge, int *weight, int *cost, int *modify) {
    // Note: the kernel does not need to know the origin vertices - their costs are simply set to 0 
    int fromv_rel_index = blockIdx.x + nv * threadIdx.x;
    if ( !modify[fromv_rel_index] ) return;  // kill thread if this vertex was not changed in the last pass
    int fromv_cost = cost[fromv_rel_index];  // get current cost for this vertex
    modify[fromv_rel_index] = 0;
    int edge_index_low  = vertex[blockIdx.x];      // block number is vertex number (one vertex per block)
    int edge_index_high = vertex[blockIdx.x + 1];  // edges out of a vertex are contiguous
    for (int edge_index = edge_index_low; edge_index < edge_index_high; edge_index++) {
        int new_cost = fromv_cost + weight[edge_index];
        int tov_rel_index = edge[edge_index] + nv * threadIdx.x;
        if (new_cost < atomicMin(cost + tov_rel_index, new_cost)) { // atomicMin returns old value
            modify[tov_rel_index] = 1; // enqueue the modified vertex for the next round
        }
    }
}

/*
// accumulate forces proportional to embedding error
// (each block should work on blockdim.x different origins, randomly)
__global__ void force (float *coord, float *force, int *cost) {
    int tindex = blockIdx.x + blockDim.x * threadIdx.x; // damn fortran ordering
    int tdindex = tindex * D;
    float dist = 0;
    float vector[D];
    for (int d = 0; d < D; d++) {
        vector[d] = (coord[tdindex + d] - something);
        dist += abs(vector[d]); // l1 norm
    }
    if (dist == 0) return; // avoid division by zero when points are superimposed
    float adjust = cost[tindex] / dist - 1;
    for (int d = 0; d < D; d++) force[tdindex + d] += adjust * vector[d];    
}

// shift embedded points according to forces, then reset forces
__global__ void integrate (float *coord, float *force) {
    int tdindex = D * (blockIdx.x + blockDim.x * threadIdx.x); // damn fortran ordering
    for (int i = tdindex; i < tdindex + D; i++) {
        coord[i] += force[i] / blockDim.x; // push points around
        force[i] = 0; // reset force to zero
    }
}
*/
